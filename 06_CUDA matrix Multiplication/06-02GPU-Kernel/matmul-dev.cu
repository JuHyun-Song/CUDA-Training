
#include <hip/hip_runtime.h>
#include <cstdio>
#include <iostream>

//Kernel program for the device(GPU): compiled by NVCC

__global__ void mulKernel(int*c, const int* a, const int*b,const int WIDTH){
	int x = threadIdx.x;
	int y = threadIdx.y;
	int i = y * WIDTH + x; //[y][x] = y * WIDTH + x;
	int sum = 0;
	for(int k = 0; k < WIDTH; ++k){
		sum += a[y * WIDTH + k] * b[k * WIDTH + x];
	}
	c[i] = sum;
}


void cpuCode(){

    //host - side data
	const int WIDTH = 5;
	int a[WIDTH][WIDTH];
	int b[WIDTH][WIDTH];
	int c[WIDTH][WIDTH] = {0};
	
	//make matrices A,B
	for(int y = 0; y < WIDTH; ++y){
		for(int x = 0; x < WIDTH; ++x){
			a[y][x] = y + x;
			b[y][x] = y + x;
		}
	}
	
	//calculation code
	for(int y=0; y < WIDTH; ++y){
		for(int x = 0; x < WIDTH; ++x){
			int sum = 0;
			for (int k = 0; k < WIDTH ; ++k){
				sum += a[y][k]*b[k][x];
			}
			c[y][x] = sum;
		}
	}
	
	//print the result
	for(int y = 0; y < WIDTH ; ++y){
		for(int x = 0 ; x < WIDTH ; ++x){
			printf("%5d",c[y][x]);
		}
		printf("\n");
	}

}

void cudaCode(){

    //host-side data
	const int WIDTH = 5;
	int a[WIDTH][WIDTH];
	int b[WIDTH][WIDTH];
	int c[WIDTH][WIDTH] = { 0 };
	
	//make a ,b matrices
	for(int y = 0; y < WIDTH; ++y){
		for(int x = 0; x < WIDTH; ++x){
			a[y][x] = y + x;
			b[y][x] = y + x;
		}
	}
	
	//allocate memory on the device
	//device-side data
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	
	//allocate device memory
	hipMalloc((void**)&dev_a, WIDTH * WIDTH * sizeof(int));
	hipMalloc((void**)&dev_b, WIDTH * WIDTH * sizeof(int));
	hipMalloc((void**)&dev_c, WIDTH*WIDTH * sizeof(int));
	
	//copy form host to device
	hipMemcpy(dev_a,a,WIDTH * WIDTH * sizeof(int),hipMemcpyHostToDevice); //dev_a = a;
	hipMemcpy(dev_b, b,WIDTH * WIDTH * sizeof(int),hipMemcpyHostToDevice); // dev_b = b;
	
	//launcn a kernel on the GPU with one thread for each element
	dim3 dimBlock(WIDTH , WIDTH, 1 ); // x, y, z
	mulKernel<<<1 , dimBlock>>> (dev_c, dev_a, dev_b, WIDTH);
	//CUDA_CHECK(cudaPeekAtLastError());

	//copy from device to host
	hipMemcpy(c,dev_c,WIDTH * WIDTH * sizeof(int),hipMemcpyDeviceToHost); //c = dev_c;

	//free device memory
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);
	
	//print the result
	for(int y = 0; y < WIDTH ; ++y){
		for(int x = 0 ; x < WIDTH ; ++x){
			printf("%d ",c[y][x]);
		}
		printf("\n");
	}

}




int main(void){

    cpuCode();
    std::cout << "+++++"<<std::endl;
    cudaCode();
	
	
	
	return 0;
}



