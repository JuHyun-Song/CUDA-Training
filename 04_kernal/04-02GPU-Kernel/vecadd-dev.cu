
#include <hip/hip_runtime.h>
#include <cstdio>
#include <stdio.h>

//kernel program for the device (GPU): compiled by NVcc
__global__ void addKernel(int*c, const int *a, const int *b){
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}
#define SIZE 1000000

int main(void){//create 

    int *a = new int[SIZE]; 
    int *b = new int[SIZE];
    int *c = new int[SIZE];
    
    for(int postion = 0 ; postion < SIZE; postion++){
        a[postion] = 2;
        b[postion] = 2;
    }
    
    //device-side data
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    
    //allocate device memory
    hipMalloc((void**)&dev_a, SIZE * sizeof(int));
    hipMalloc((void**)&dev_b, SIZE * sizeof(int));
    hipMalloc((void**)&dev_c, SIZE * sizeof(int));

    //copy from host to device
    hipMemcpy(dev_a, a, SIZE * sizeof(int), hipMemcpyHostToDevice);// dev_a = a;
    hipMemcpy(dev_b, b, SIZE * sizeof(int), hipMemcpyHostToDevice);// dev_b = b;
    hipMemcpy(dev_c, b, SIZE * sizeof(int), hipMemcpyHostToDevice);// dev_b = b;
    
    //launch a Kernel on the GPU with one thread for each element.
    addKernel<<<1,SIZE>>>(dev_c, dev_a, dev_b);

    //copy from device to host
    hipMemcpy(c, dev_c, SIZE * sizeof(int), hipMemcpyDeviceToHost);// c = dev_c;

    //free device memory
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    //print the result
    printf("{%d,%d,%d,%d,%d} + {%d,%d,%d,%d,%d}"
    "={%d,%d,%d,%d,%d}\n",
    a[0],a[1],a[2],a[3],a[4],
    b[0],b[1],b[2],b[3],b[4],
    c[0],c[1],c[2],c[3],c[4]);

    return 0;
}

