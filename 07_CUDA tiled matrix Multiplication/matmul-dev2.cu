
#include <hip/hip_runtime.h>
#include <cstdio>
#include <stdlib.h> //for rand(),malloc(),free()


const int WIDTH = 1024; // total width is 1024*1024
const int TILE_WIDTH = 32; //block will be(TILE_WIDTH,TILE_WIDTH)
constexpr int GRID_WIDTH = 128;//(WIDTH / TILE_WIDTH); //grid will be (GRID_WIDTH,GRID_WIDTH)

__global__ void matmaul(float*c, const float*a, const float*b, const int width){
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	float sum = 0.0F;
	for(register int k = 0; k < width; ++k){
		float lhs = a[y * width + k];
		float rhs = b[k * width + x];
		sum += lhs * rhs;
	}
	c[y * width + x] = sum;
}

void genData(float* ptr, unsigned int size){
	for(int i = 0 ; i < size; i++){
		*ptr++ =(float)(rand()%1000) / 1000.0F;
	}
}

int main(void){
	
	
	printf("23");
	float** a = new float*[WIDTH];
    float** b = new float*[WIDTH];
    float** c = new float*[WIDTH];
    
	printf("23");
    for(int i = 0; i < WIDTH; ++i){
        a[i] = new float[WIDTH];
        b[i] = new float[WIDTH];
        c[i] = new float[WIDTH];
	}
	printf("23");
	//generate source data
	genData(&(a[0][0]),WIDTH * WIDTH);
	genData(&(b[0][0]),WIDTH * WIDTH);
	
	//device-side data
	float* dev_a = 0;
	float* dev_b = 0;
	float* dev_c = 0;
	printf("23");
	
	//allocate device memory	
	hipMalloc((void**)&dev_a,WIDTH*WIDTH*sizeof(float));
	hipMalloc((void**)&dev_b,WIDTH*WIDTH*sizeof(float));
	hipMalloc((void**)&dev_c,WIDTH*WIDTH*sizeof(float));

	//copy from host to device
	hipMemcpy(dev_a, a, WIDTH*WIDTH*sizeof(float), hipMemcpyHostToDevice); // dev_a = a;
	hipMemcpy(dev_b, b, WIDTH*WIDTH*sizeof(float), hipMemcpyHostToDevice); // dev_a = a;

	//CUDA:launch the kernel
	dim3 dimGrid(GRID_WIDTH, GRID_WIDTH, 1);
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
	matmaul<<<dimGrid,dimBlock>>>(dev_c,dev_a,dev_b,WIDTH);

	
	// copy from device to host
	hipMemcpy(c, dev_c, WIDTH * WIDTH * sizeof(float),hipMemcpyDeviceToHost); // c = dev_c;

	//free device memory
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);
	
	//print the result
	
	for(int y = 0; y < WIDTH; ++y){
	
		for(int x = 0; x < WIDTH; ++x){
		
			printf("%5f ", c[y][x]);
		}
		printf("\n");
	}
	
	
	return 0;
}



